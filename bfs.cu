//BFS using vertexAPI2

#include "util.h"
#include "graphio.h"
#include "refgas.h"
#include "gpugas.h"


//nvcc doesn't like the __device__ variable to be a static member inside BFS
//so these are both outside.
int g_iterationCount;
__device__ __constant__ int g_iterationCountGPU;


struct BFS
{
  struct VertexData
  {
    int depth;
  };

  struct EdgeData {}; //nothing

  typedef int GatherResult;
  static const int gatherZero = INT_MAX - 1;

  __host__ __device__
  static int gatherReduce(const int& left, const int& right)
  {
    return 0; //do nothing
  }


  __host__ __device__
  static int gatherMap(const VertexData* dst, const VertexData *src, const EdgeData* edge)
  {
    return 0; //do nothing
  }


  __host__ __device__
  static bool apply(VertexData* vert, int dist)
  {
    if( vert->depth == -1 )
    {
      #ifdef __CUDA_ARCH__
        vert->depth = g_iterationCountGPU;
      #else
        vert->depth = g_iterationCount;
      #endif        
      return true;
    }
    return false;
  }


  __host__ __device__
  static void scatter(const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


template<bool GPU>
void setIterationCount(int v)
{
  if( GPU )
    hipMemcpyToSymbol(HIP_SYMBOL(g_iterationCountGPU), &v, sizeof(v));
  else
    g_iterationCount = v;
}


template<typename Engine, bool GPU>
void run(int nVertices, BFS::VertexData* vertexData, int nEdges
  , const int *srcs, const int *dsts, int sourceVertex)
{
  Engine engine;
  engine.setGraph(nVertices, vertexData, nEdges, 0, &srcs[0], &dsts[0]);
  engine.setActive(sourceVertex, sourceVertex+1);
  int iter = 0;
  setIterationCount<GPU>(iter);  
  while( engine.countActive() )
  {
    //run apply without gather
    engine.gatherApply(false);
    engine.scatterActivate(false);
    engine.nextIter();
    setIterationCount<GPU>(++iter);
  }
  engine.getResults();
}


int main(int argc, char** argv)
{
  char *inputFilename;
  int sourceVertex;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "si-t-d", &inputFilename, &sourceVertex
    , &runTest, &dumpResults) )
  {
    printf("Usage: bfs [-t] [-d] inputfile source\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  loadGraph(inputFilename, nVertices, srcs, dsts);

  //initialize vertex data
  std::vector<BFS::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].depth = -1; 

  std::vector<BFS::VertexData> refVertexData;
  if( runTest )
  {
    refVertexData = vertexData;
    run<GASEngineRef<BFS>, false>(nVertices, &refVertexData[0], (int)srcs.size()
      , &srcs[0], &dsts[0], sourceVertex);
    if( dumpResults )
    {
      printf("Reference:\n");
      for( int i = 0; i < nVertices; ++i )
        printf("%d %d\n", i, refVertexData[i].depth);
    }
  }

  run<GASEngineGPU<BFS>, true>(nVertices, &vertexData[0], (int) srcs.size()
    , &srcs[0], &dsts[0], sourceVertex);
  if( dumpResults )
  {
    printf("GPU:\n");
    for( int i = 0; i < nVertices; ++i )
      printf("%d %d\n", i, vertexData[i].depth);
  }

  if( runTest )
  {
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( vertexData[i].depth != refVertexData[i].depth )
      {
        printf("%d %d %d\n", i, refVertexData[i].depth, vertexData[i].depth);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  return 0;
}
