#include "hip/hip_runtime.h"
#include "refgas.h"
#include "gpugas.h"
#include "util.h"
#include "graphio.h"
#include <vector>


//Vertex program for Pagerank
struct PageRank
{  
  static const float pageConst = 0.15f;
  static const float tol = 0.01f;
    
  struct VertexData
  {
    float rank;
    int   numOutEdges;
  };

  struct EdgeData {};

  typedef float GatherResult;

  static const float gatherZero = 0.0f;

  __host__ __device__
  static float gatherMap(const VertexData* dst, const VertexData* src, const EdgeData* edge)
  {
    //this division is being done too many times right?
    //should just store the normalized value in apply?
    return src->rank / src->numOutEdges;
  }

  __host__ __device__
  static float gatherReduce(const float& left, const float& right)
  {
    return left + right;
  }

  __host__ __device__
  static bool apply(VertexData* vertexData, const float& gatherResult)
  {
    float newRank = pageConst + (1.0f - pageConst) * gatherResult;
    bool ret = fabs(newRank - vertexData->rank) >= tol;
    vertexData->rank = newRank;
    return ret;
  }

  __host__ __device__
  static void scatter(const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


void outputRanks(int n, const PageRank::VertexData* vertexData)
{
  for( int i = 0; i < n; ++i )
  {
    printf("%d %f\n", i, vertexData[i].rank);
  }
}


template<typename Engine>
void run(int nVertices, PageRank::VertexData* vertexData, int nEdges
  , const int* srcs, const int* dsts)
{
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].rank = PageRank::pageConst;

  Engine engine;
  engine.setGraph(nVertices, vertexData, nEdges, 0, srcs, dsts);
  //all vertices begin active for pagerank
  engine.setActive(0, nVertices);
  engine.run();
  engine.getResults();
}


int main(int argc, char **argv)
{
  char* inputFilename;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "s-t-d", &inputFilename, &runTest, &dumpResults) )
  {
    printf("Usage: pagerank [-t] [-d] inputfile\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  loadGraph(inputFilename, nVertices, srcs, dsts);
  printf("loaded %s with %d vertices and %zd edges\n", inputFilename, nVertices, srcs.size());

  //initialize vertex data
  //convert to CSR to get the count of edges.
  std::vector<int> srcOffsets(nVertices + 1);
  std::vector<int> csrSrcs(srcs.size());
  edgeListToCSR<int>(nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);
  
  std::vector<PageRank::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].numOutEdges = srcOffsets[i + 1] - srcOffsets[i];

  std::vector<PageRank::VertexData> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    run< GASEngineRef<PageRank> >(nVertices, &refVertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
    if( dumpResults )
    {
      printf("Reference\n");
      outputRanks(nVertices, &refVertexData[0]);
    }
  }

  run< GASEngineGPU<PageRank> >(nVertices, &vertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
  if( dumpResults )
  {
    printf("GPU:\n");
    outputRanks(nVertices, &vertexData[0]);
  }

  if( runTest )
  {
    const float tol = 1.0e-6f;
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( fabs(vertexData[i].rank - refVertexData[i].rank) > tol )
      {
        printf("%d %f %f\n", i, refVertexData[i].rank, vertexData[i].rank);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }
  
  return 0;
}

