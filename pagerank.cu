#include "hip/hip_runtime.h"
/******************************************************************************
Copyright 2013 Royal Caliber LLC. (http://www.royal-caliber.com)

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
******************************************************************************/

#include "refgas.h"
#include "gpugas.h"
#include "util.h"
#include "graphio.h"
#include <vector>
#include <iostream>


//Vertex program for Pagerank
struct PageRank
{  
  static const float pageConst = 0.15f;
  static const float tol = 0.01f;
    
  struct VertexData
  {
    float rank;
    int   numOutEdges;
    friend std::ostream& operator<<(std::ostream &out, const VertexData &data);
  };

  struct EdgeData {};

  typedef float GatherResult;

  static const float gatherZero = 0.0f;

  enum { Commutative = true };

  __host__ __device__
  static float gatherMap(const VertexData* dst, const VertexData* src, const EdgeData* edge)
  {
    //this division is being done too many times right?
    //should just store the normalized value in apply?
    return src->rank / src->numOutEdges;
  }

  __host__ __device__
  static float gatherReduce(const float& left, const float& right)
  {
    return left + right;
  }

  __host__ __device__
  static bool apply(VertexData* vertexData, const float& gatherResult)
  {
    float newRank = pageConst + (1.0f - pageConst) * gatherResult;
    bool ret = fabs(newRank - vertexData->rank) >= tol;
    vertexData->rank = newRank;
    return ret;
  }

  __host__ __device__
  static void scatter(const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


void outputRanks(int n, const PageRank::VertexData* vertexData, FILE* f = stdout)
{
  for( int i = 0; i < n; ++i )
  {
    fprintf(f, "%d %f\n", i, vertexData[i].rank);
  }
}


template<typename Engine>
int64_t run(int nVertices, PageRank::VertexData* vertexData, int nEdges
  , const int* srcs, const int* dsts)
{
  Engine engine;
  #ifdef VERTEXAPI_USE_MPI
    engine.initMPI();
  #endif
  engine.setGraph(nVertices, vertexData, nEdges, 0, srcs, dsts);
  //all vertices begin active for pagerank
  engine.setActive(0, nVertices);
  int64_t t0 = currentTime();
  engine.run();
  engine.getResults();
  int64_t t1 = currentTime();
  return t1 - t0;
}


int main(int argc, char **argv)
{
  #ifdef VERTEXAPI_USE_MPI
    int mpiRank = 0;
    int mpiSize = 0; //number of mpi nodes

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpiRank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpiSize);
  #endif

  #ifdef VERTEXAPI_USE_MPI
    #define MASTER (mpiRank == 0)
  #else
    #define MASTER (1)
  #endif
  
  char* inputFilename;
  char* inputDegreeFilename;
  char* outputFilename = 0;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "ss-t-d|s"
    , &inputFilename, &inputDegreeFilename, &runTest, &dumpResults, &outputFilename) )
  {
    printf("Usage: pagerank [-t] [-d] inputEdges inputDegrees [outputfile]\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  #ifdef VERTEXAPI_USE_MPI
    std::string tmp;
    tmp = filenameSuffixMPI(inputFilename, mpiRank, mpiSize);
    loadGraph(tmp.c_str(), nVertices, srcs, dsts);
  #else
    loadGraph(inputFilename, nVertices, srcs, dsts);
  #endif  
  printf("loaded %s with %d vertices and %zd edges\n", inputFilename, nVertices, srcs.size());

  //read in the out-degree for the vertices
  //we are also using this to infer the total number of vertices
  std::vector<int> outDegrees;
  loadData(inputDegreeFilename, outDegrees);
  nVertices = outDegrees.size();
  
  std::vector<PageRank::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
  {
    vertexData[i].numOutEdges = outDegrees[i];
    vertexData[i].rank = PageRank::pageConst;
  }
  
  std::vector<PageRank::VertexData> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    run< GASEngineRef<PageRank> >(nVertices, &refVertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
    if( MASTER && dumpResults )
    {
      printf("Reference\n");
      outputRanks(nVertices, &refVertexData[0]);
    }
  }

  int64_t t = run< GASEngineGPU<PageRank> >(nVertices, &vertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
  
  if( MASTER )
    printf("Took %f ms\n", t/1000.0f);
    
  if( MASTER && dumpResults )
  {
    printf("GPU:\n");
    outputRanks(nVertices, &vertexData[0]);
  }

  if( MASTER && runTest )
  {
    const float tol = 1.0e-6f;
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( fabs(vertexData[i].rank - refVertexData[i].rank) > tol )
      {
        printf("%d %f %f\n", i, refVertexData[i].rank, vertexData[i].rank);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  if( MASTER && outputFilename )
  {
    FILE* f = fopen(outputFilename, "w");
    printf("writing results to file %s\n", outputFilename);
    outputRanks(nVertices, &vertexData[0], f);
    fclose(f);
  }

  free(inputFilename);
  free(outputFilename);

  #ifdef VERTEXAPI_USE_MPI
    MPI_Finalize();
  #endif
  
  return 0;
}

std::ostream& operator<<(std::ostream &out, const PageRank::VertexData &data) {
  out << data.rank;
  return out;
}
