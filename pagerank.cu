#include "hip/hip_runtime.h"
/******************************************************************************
Copyright 2013 Royal Caliber LLC. (http://www.royal-caliber.com)

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
******************************************************************************/

//this should come from the Makefile, putting it in here for testing
#define VERTEXAPI_USE_MPI

#include "refgas.h"
#include "gpugas.h"
#include "util.h"
#include "graphio.h"
#include <vector>


//Vertex program for Pagerank
struct PageRank
{  
  static const float pageConst = 0.15f;
  static const float tol = 0.01f;
    
  struct VertexData
  {
    float rank;
    int   numOutEdges;
  };

  struct EdgeData {};

  typedef float GatherResult;

  static const float gatherZero = 0.0f;

  enum { Commutative = true };

  __host__ __device__
  static float gatherMap(const VertexData* dst, const VertexData* src, const EdgeData* edge)
  {
    //this division is being done too many times right?
    //should just store the normalized value in apply?
    return src->rank / src->numOutEdges;
  }

  __host__ __device__
  static float gatherReduce(const float& left, const float& right)
  {
    return left + right;
  }

  __host__ __device__
  static bool apply(VertexData* vertexData, const float& gatherResult)
  {
    float newRank = pageConst + (1.0f - pageConst) * gatherResult;
    bool ret = fabs(newRank - vertexData->rank) >= tol;
    vertexData->rank = newRank;
    return ret;
  }

  __host__ __device__
  static void scatter(const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


void outputRanks(int n, const PageRank::VertexData* vertexData, FILE* f = stdout)
{
  for( int i = 0; i < n; ++i )
  {
    fprintf(f, "%d %f\n", i, vertexData[i].rank);
  }
}


template<typename Engine>
void run(int nVertices, PageRank::VertexData* vertexData, int nEdges
  , const int* srcs, const int* dsts)
{
  Engine engine;
  engine.setGraph(nVertices, vertexData, nEdges, 0, srcs, dsts);
  #ifdef VERTEXAPI_USE_MPI
    engine.initMPI();
  #endif
  //all vertices begin active for pagerank
  engine.setActive(0, nVertices);
  int64_t t0 = currentTime();
  engine.run();
  engine.getResults();
  int64_t t1 = currentTime();
  printf("Took %f ms\n", (t1 - t0)/1000.0f);
}


int main(int argc, char **argv)
{
  int mpiRank = 0;

  #ifdef VERTEXAPI_USE_MPI
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpiRank);
  #endif
  
  char* inputFilename;
  char* inputDegreeFilename;
  char* outputFilename = 0;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "ss-t-d|s"
    , &inputFilename, &inputDegreeFilename, &runTest, &dumpResults, &outputFilename) )
  {
    printf("Usage: pagerank [-t] [-d] inputEdges inputDegrees [outputfile]\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  #ifdef VERTEXAPI_USE_MPI
    char rankStr[8];
    snprintf(rankStr, sizeof(rankStr), "_%d", mpiRank);
    std::string tmp = inputFilename;
    tmp += rankStr;
    loadGraph(tmp.c_str(), nVertices, srcs, dsts);
    printf("%d: loaded %zd edges\n", mpiRank, srcs.size());
  #else
    loadGraph(inputFilename, nVertices, srcs, dsts);
    printf("loaded %s with %d vertices and %zd edges\n", inputFilename, nVertices, srcs.size());
  #endif

  //read in the out-degree for the vertices
  std::vector<int> outDegrees;
  loadData(inputDegreeFilename, outDegrees);

  //Get the actual number of vertices
  #ifdef VERTEXAPI_USE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Allreduce(MPI_IN_PLACE, &nVertices, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
    printf("%d: nVertices = %d\n", mpiRank, nVertices);
  #endif

  //initialize vertex data
  //convert to CSR to get the count of edges.
  std::vector<int> srcOffsets(nVertices + 1);
  std::vector<int> csrSrcs(srcs.size());
  edgeListToCSR<int>(nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);
  
  std::vector<PageRank::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
  {
    vertexData[i].numOutEdges = outDegrees[i]; //srcOffsets[i + 1] - srcOffsets[i];
    vertexData[i].rank = PageRank::pageConst;
  }
  
  std::vector<PageRank::VertexData> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    run< GASEngineRef<PageRank> >(nVertices, &refVertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
    if( dumpResults )
    {
      printf("Reference\n");
      outputRanks(nVertices, &refVertexData[0]);
    }
  }

//  run< GASEngineGPU<PageRank> >(nVertices, &vertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
//  if( dumpResults )
//  {
//    printf("GPU:\n");
//    outputRanks(nVertices, &vertexData[0]);
//  }

  if( runTest )
  {
    const float tol = 1.0e-6f;
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( fabs(vertexData[i].rank - refVertexData[i].rank) > tol )
      {
        printf("%d %f %f\n", i, refVertexData[i].rank, vertexData[i].rank);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  if( outputFilename && mpiRank == 0 )
  {
    FILE* f = fopen(outputFilename, "w");
    printf("writing results to file %s\n", outputFilename);
    outputRanks(nVertices, &vertexData[0], f);
    fclose(f);
  }

  free(inputFilename);
  free(outputFilename);

  #ifdef VERTEXAPI_USE_MPI
    MPI_Finalize();
  #endif
  
  return 0;
}

