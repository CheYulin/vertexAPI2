#include "hip/hip_runtime.h"
//Single source shortest paths using vertexAPI2

#include "util.h"
#include "graphio.h"
#include "refgas.h"
#include "gpugas.h"
#include <climits>

struct SSSP
{
  //making these typedefs rather than singleton structs
  typedef int VertexData;
  typedef int EdgeData;

  typedef int GatherResult;
  static const int maxLength = 100000;
  static const int gatherZero = INT_MAX - maxLength;


  __host__ __device__
  static int gatherReduce(const int& left, const int& right)
  {
    return min(left, right);
  }


  __host__ __device__
  static int gatherMap(const VertexData* dstDist, const VertexData *srcDist, const EdgeData* edgeLen)
  {
    return *srcDist + *edgeLen;
  }


  __host__ __device__
  static bool apply(VertexData* curDist, GatherResult dist)
  {
    bool changed = dist < *curDist;
    *curDist = min(*curDist, dist);
    return changed;
  }


  __host__ __device__
  static void scatter(const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


template<typename Engine>
void run(int nVertices, SSSP::VertexData* vertexData, int nEdges
  , SSSP::EdgeData* edgeData, const int* srcs, const int* dsts)
{
    Engine engine;
    engine.setGraph(nVertices, vertexData, nEdges, edgeData, srcs, dsts);

    //TODO, setting all vertices to active for first step works, but it would
    //be faster to instead set to neighbors of starting vertex
    engine.setActive(0, nVertices);
    engine.run();
    engine.getResults();
}


int main(int argc, char** argv)
{
  char *inputFilename;
  int sourceVertex;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "si-t-d", &inputFilename, &sourceVertex
    , &runTest, &dumpResults) )
  {
    printf("Usage: sssp [-t] [-d] inputfile source\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  std::vector<int> edgeData;
  loadGraph(inputFilename, nVertices, srcs, dsts, &edgeData);
  if( edgeData.size() == 0 )
  {
    printf("No edge data available in input file\n");
    exit(1);
  }

  //initialize vertex data
  std::vector<int> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i] = SSSP::gatherZero;
  vertexData[sourceVertex] = 0;

  std::vector<int> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    run< GASEngineRef<SSSP> >(nVertices, &refVertexData[0], (int)srcs.size()
      , &edgeData[0], &srcs[0], &dsts[0]);
    if( dumpResults )
    {
      printf("Reference\n");
      for (int i = 0; i < nVertices; ++i)
        printf("%d %d\n", i, refVertexData[i]);
    }  
  }

  run< GASEngineGPU<SSSP> >(nVertices, &vertexData[0], (int)srcs.size()
    , &edgeData[0], &srcs[0], &dsts[0]);
  if( dumpResults )
  {
    for (int i = 0; i < nVertices; ++i)
      printf("%d %d\n", i, vertexData[i]);
  }

  if( runTest )
  {
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( vertexData[i] != refVertexData[i] )
      {
        printf("%d %d %d\n", i, refVertexData[i], vertexData[i]);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  return 0;
}
